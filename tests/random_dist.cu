#include "hip/hip_runtime.h"
#include "mpicufftslabs.hpp"
#include ""
#include "cufft.hpp"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <unistd.h>
#include <iostream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error at %s:%d\n",__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)

#define Nx 32
#define Ny 32
#define Nz 32

#define CUDA_AWARE 0

using R_t = typename cuFFT<float>::R_t;
using C_t = typename cuFFT<float>::C_t;

__global__ void scaleUniformArray(R_t* data_d, R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

__global__ void difference(C_t* array1, C_t* array2, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        array1[i].x -= array2[i].x;
        array1[i].y -= array2[i].y;
    }
}


int initializeRandArray(void* in_d){
    hiprandGenerator_t gen;
    R_t *real = cuFFT<float>::real(in_d);

    //create pseudo-random generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    //set seed of generator
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    //get poisson samples
    CURAND_CALL(hiprandGenerateUniform(gen, real, Nx*Ny*Nz));

    scaleUniformArray<<<(Nx*Ny*Nz)/1024+1, 1024>>>(real, 255, Nx*Ny*Nz);

    return 0;
}

int main() {      
    //initialize MPI
    MPI_Init(NULL, NULL);

    std::vector<MPI_Request> send_req;
    std::vector<MPI_Request> recv_req;

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    world_size--;

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == world_size){
        R_t *in_d, *send_ptr;
        C_t *out_d, *recv_ptr, *res_d;

        size_t ws_r2c;

        hipfftHandle planR2C;
        hipblasHandle_t handle;

        send_req.resize(world_size, MPI_REQUEST_NULL);
        recv_req.resize(world_size, MPI_REQUEST_NULL);

        //allocate memory (device)
        CUDA_CALL(hipMalloc((void **)&in_d, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipMalloc((void **)&out_d, Nx*Ny*Nz*sizeof(C_t)));
        
        if (CUDA_AWARE == 1){
            CUDA_CALL(hipMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
            CUDA_CALL(hipMalloc((void **)&recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
            res_d = recv_ptr;
        } else {
            CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
            CUDA_CALL(hipHostMalloc((void **)&recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
            CUDA_CALL(hipMalloc((void **)&res_d, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
        }

        std::cout << "Initialize Random Array" << std::endl;
        initializeRandArray(in_d);

        CUDA_CALL(hipMemcpyAsync(send_ptr, in_d, Nx*Ny*Nz*sizeof(R_t), 
            CUDA_AWARE==1?hipMemcpyDeviceToDevice:hipMemcpyDeviceToHost));

        CUFFT_CALL(hipfftCreate(&planR2C));
        CUFFT_CALL(hipfftSetAutoAllocation(planR2C, 0));
        CUFFT_CALL(hipfftMakePlan3d(planR2C, Nx, Ny, Nz, cuFFT<float>::R2Ctype, &ws_r2c));

        CUDA_CALL(hipDeviceSynchronize());
        CUFFT_CALL(hipfftSetWorkArea(planR2C, in_d));

        std::cout << "Distribute Array" << std::endl;
        size_t N1 = Nx/world_size;
        size_t N2 = Ny/world_size;
        size_t send_count = 0;
        size_t recv_count = 0;
        for (int pidx = 0; pidx < world_size; pidx++){
            size_t Nxpidx = N1 + (pidx<Nx%world_size?1:0);
            size_t Nypidx = N2 + (pidx<Ny%world_size?1:0);
            recv_req[pidx] = MPI_REQUEST_NULL;
            send_req[pidx] = MPI_REQUEST_NULL;

            std::cout << pidx << ": distribute with recv_count=" << recv_count << ", send_count=" << send_count << std::endl;
            std::cout << pidx << ": distribute with next recv_count=" << recv_count + Nx*Nypidx*(Nz/2+1) << ", next send_count=" << send_count + Nxpidx*Ny*Nz << std::endl;

            MPI_Irecv(&recv_ptr[recv_count], Nx*Nypidx*(Nz/2+1)*sizeof(C_t), MPI_BYTE, pidx, pidx, MPI_COMM_WORLD, &recv_req[pidx]);
            recv_count += Nx*Nypidx*(Nz/2+1);

            MPI_Isend(&send_ptr[send_count], Nxpidx*Ny*Nz*sizeof(R_t), MPI_BYTE, pidx, pidx, MPI_COMM_WORLD, &send_req[pidx]);
            send_count += Nxpidx*Ny*Nz;
        }

        std::cout << "Wait" << std::endl;
        MPI_Waitall(world_size, send_req.data(), MPI_STATUSES_IGNORE);
        std::cout << "Completed Send Wait Period" << std::endl;


        R_t *real    = cuFFT<float>::real(in_d);
        C_t *complex = cuFFT<float>::complex(out_d);

        CUFFT_CALL(cuFFT<float>::execR2C(planR2C, real, complex));
        CUDA_CALL(hipDeviceSynchronize());

        CUBLAS_CALL(hipblasCreate(&handle));


        MPI_Waitall(world_size, recv_req.data(), MPI_STATUSES_IGNORE);

        std::cout << "Completed Wait Period" << std::endl;

        if (CUDA_AWARE==0){
            CUDA_CALL(hipMemcpyAsync(res_d, recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipDeviceSynchronize());
        }

        std::cout << "Compute Difference" << std::endl;
        // difference<<<(Nx*Ny*(Nz/2+1))/1024+1, 1024>>>(complex, res_d, Nx*Ny*(Nz/2+1));
        float sum = 0;
        CUBLAS_CALL(hipblasScasum(handle, Nx*Ny*(Nz/2+1), complex, 1, &sum));
        CUBLAS_CALL(hipblasDestroy(handle));

        std::cout << "Result " << sum << std::endl;

        CUFFT_CALL(hipfftDestroy(planR2C));

        CUDA_CALL(hipFree(in_d));
        CUDA_CALL(hipFree(out_d));
        if (CUDA_AWARE == 0){
            CUDA_CALL(hipFree(res_d));
        } 
    } else{
        size_t N1=Nx/world_size;
        size_t N2=Ny/world_size;
        if (rank < Nx%world_size)
            N1++;
        if (rank < Ny%world_size)
            N2++;
    
        send_req.resize(1, MPI_REQUEST_NULL);
        recv_req.resize(1, MPI_REQUEST_NULL);

        R_t *in_d, *recv_ptr;
        C_t *out_d, *send_ptr;
        size_t out_size = std::max(N1*Ny*(Nz/2+1), Nx*N2*(Nz/2+1));

        //allocate memory (device)
        CUDA_CALL(hipMalloc((void **)&in_d, N1*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipMalloc((void **)&out_d, out_size*sizeof(C_t)));
        
        if (CUDA_AWARE == 1){
            recv_ptr = in_d;
            send_ptr = out_d;
        } else {
            CUDA_CALL(hipHostMalloc((void **)&recv_ptr, N1*Ny*Nz*sizeof(R_t)));
            CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*N2*(Nz/2+1)*sizeof(C_t)));
        }
        
        std::cout << "1: Wait recv" << std::endl;

        //receive input data via MPI
        MPI_Irecv(recv_ptr, N1*Ny*Nz*sizeof(R_t), MPI_BYTE, world_size, rank, MPI_COMM_WORLD, &recv_req[0]);
        MPI_Wait(&recv_req[0], MPI_STATUSES_IGNORE);

        std::cout << "1: Wait completed" << std::endl;

        if (CUDA_AWARE == 0){
            CUDA_CALL(hipMemcpyAsync(recv_ptr, in_d, N1*Ny*Nz*sizeof(R_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipDeviceSynchronize());
        }

        std::cout << "1: Run Distributed FFT" << std::endl;

        //initialize MPIcuFFT
        MPIcuFFT_Slabs<float> mpicuFFT(MPI_COMM_WORLD, CUDA_AWARE==1);
        mpicuFFT.initFFT(Nx, Ny, Nz, true);
    
        //execute
        mpicuFFT.execR2C(out_d, in_d);
        hipblasHandle_t handle;
        CUBLAS_CALL(hipblasCreate(&handle));
        float sum = 0;
        CUBLAS_CALL(hipblasScasum(handle, Nx*Ny*(Nz/2+1), out_d, 1, &sum));
        CUBLAS_CALL(hipblasDestroy(handle));

        std::cout << "1: Distributed FFT complete " << sum << std::endl;

        if (CUDA_AWARE == 0){
            CUDA_CALL(hipMemcpyAsync(send_ptr, out_d, Nx*N2*(Nz/2+1)*sizeof(C_t), hipMemcpyDeviceToHost));
            CUDA_CALL(hipDeviceSynchronize());
        }

        MPI_Isend(send_ptr, Nx*N2*(Nz/2+1)*sizeof(C_t), MPI_BYTE, world_size, rank, MPI_COMM_WORLD, &send_req[0]);

        std::cout << "1: Wait send" << std::endl;

        MPI_Wait(&send_req[0], MPI_STATUSES_IGNORE);

        std::cout << "1: Wait complete" << std::endl;

        
        CUDA_CALL(hipFree(in_d));
        CUDA_CALL(hipFree(out_d));
    }
    
    //finalize
    MPI_Finalize();
    return 0;
}

