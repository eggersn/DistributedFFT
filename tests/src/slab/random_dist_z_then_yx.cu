#include "hip/hip_runtime.h"
#include "mpicufft_slab_z_then_yx.hpp"
#include "mpicufft_slab_z_then_yx_opt1.hpp"
#include "tests_slab_random_z_then_yx.hpp"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error at %s:%d\n",__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {             \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)

namespace Difference_Slab_Z_Then_YX {
    __global__ void differenceFloat(cuFFT<float>::C_t* array1, cuFFT<float>::C_t* array2, int n){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) {
            array1[i].x -= array2[i].x;
            array1[i].y -= array2[i].y;
        }
    }
    
    __global__ void differenceDouble(cuFFT<double>::C_t* array1, cuFFT<double>::C_t* array2, int n){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) {
            array1[i].x -= array2[i].x;
            array1[i].y -= array2[i].y;
        }
    }

    __global__ void differenceFloatInv(cuFFT<float>::R_t* array1, cuFFT<float>::R_t* array2, int n, cuFFT<float>::R_t scalar){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) {
            array1[i] -= scalar * array2[i];
        }
    }
    
    __global__ void differenceDoubleInv(cuFFT<double>::R_t* array1, cuFFT<double>::R_t* array2, int n, cuFFT<double>::R_t scalar){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) {
            array1[i] -= scalar * array2[i];
        }
    }
    
    template<typename T> 
    struct Difference { 
        static decltype(differenceFloat)* difference;
        static decltype(differenceFloatInv)* differenceInv;
    };
    template<typename T> decltype(differenceFloat)* Difference<T>::difference = differenceFloat;
    template<typename T> decltype(differenceFloatInv)* Difference<T>::differenceInv = differenceFloatInv;
    
    template<> struct Difference<double> { 
        static decltype(differenceDoubleInv)* differenceInv;
        static decltype(differenceDouble)* difference;
    };
    decltype(differenceDouble)* Difference<double>::difference = differenceDouble;    
    decltype(differenceDoubleInv)* Difference<double>::differenceInv = differenceDoubleInv;   
}

template<typename T> 
int Tests_Slab_Random_Z_Then_YX<T>::run(const int testcase, const int opt, const int runs){
    if (testcase == 0)
        return this->testcase0(opt, runs);
    else if (testcase == 1)
        return this->testcase1(opt, runs);
    else if (testcase == 2)
        return this->testcase2(opt, runs);
    return -1;
}

template<typename T> 
int Tests_Slab_Random_Z_Then_YX<T>::testcase0(const int opt, const int runs){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;
    int provided; 
    //initialize MPI
    MPI_Init_thread(NULL, NULL, MPI_THREAD_MULTIPLE, &provided);
    if (provided < MPI_THREAD_MULTIPLE) {
        printf("ERROR: The MPI library does not have full thread support\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int dev_count;
    CUDA_CALL(hipGetDeviceCount(&dev_count));
    CUDA_CALL(hipSetDevice(rank % dev_count));

    size_t N1=Nx/world_size;
    size_t N2=(Nz/2+1)/world_size;
    if (rank < Nx%world_size)
        N1++;
    if (rank < (Nz/2+1)%world_size)
        N2++;

    R_t *in_d;
    C_t *out_d;
    size_t out_size = std::max(N1*Ny*(Nz/2+1), Nx*Ny*N2);

    //allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, N1*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, out_size*sizeof(C_t)));
    //allocate memory (host)
    // out_h = (C_t *)calloc(out_size, sizeof(C_t));
    
    MPIcuFFT_Slab_Z_Then_YX<T> *mpicuFFT;
    if (opt == 1)
        mpicuFFT = new MPIcuFFT_Slab_Z_Then_YX_Opt1<T>(config, MPI_COMM_WORLD, world_size);
    else 
        mpicuFFT = new MPIcuFFT_Slab_Z_Then_YX<T>(config, MPI_COMM_WORLD, world_size);
    
    GlobalSize global_size(Nx, Ny, Nz);
    mpicuFFT->initFFT(&global_size, true);
    //execute
    for (int i = 0; i < runs; i++){
        this->initializeRandArray(in_d, N1);
        MPI_Barrier(MPI_COMM_WORLD);
        mpicuFFT->execR2C(out_d, in_d);
        MPI_Barrier(MPI_COMM_WORLD);
    }
    
    // CUDA_CALL(hipMemcpy(out_h, out_d, out_size*sizeof(C_t), hipMemcpyDeviceToHost));

    //do stuff with out_h

    //finalize
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();

    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(out_d));
    // free(out_h);

    delete mpicuFFT;
    return 0;
}

template<typename T>
int Tests_Slab_Random_Z_Then_YX<T>::testcase1(const int opt, const int runs) {      
    int provided; 
    //initialize MPI
    MPI_Init_thread(NULL, NULL, MPI_THREAD_MULTIPLE, &provided);
    if (provided < MPI_THREAD_MULTIPLE) {
        printf("ERROR: The MPI library does not have full thread support\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    world_size--;

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == world_size){
        this->coordinate(world_size, runs);
    } else{
        this->compute(rank, world_size, opt, runs);
    }
    
    //finalize
    MPI_Finalize();

    return 0;
}

template <typename T>
int Tests_Slab_Random_Z_Then_YX<T>::coordinate(const int world_size, const int runs){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    std::vector<MPI_Request> send_req;
    std::vector<MPI_Request> recv_req;

    R_t *in_d, *send_ptr;
    C_t *out_d, *recv_ptr, *res_d;

    size_t ws_r2c;

    hipfftHandle planR2C;
    hipblasHandle_t handle;

    //allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, Nx*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    CUDA_CALL(hipMalloc((void **)&res_d, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    
    if (config.cuda_aware){
        CUDA_CALL(hipMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipMalloc((void **)&recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    } else {
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    }

    MPI_Comm temp;
    MPI_Comm_split(MPI_COMM_WORLD, MPI_UNDEFINED, 0, &temp);

    for (int i = 0; i < runs; i++) {
        send_req.resize(world_size, MPI_REQUEST_NULL);
        recv_req.resize(world_size, MPI_REQUEST_NULL);
        //random initialization of full Nx*Ny*Nz array
        this->initializeRandArray(in_d, Nx);
    
        //Copy input data to send-buffer and initialize cufft
        CUDA_CALL(hipMemcpyAsync(send_ptr, in_d, Nx*Ny*Nz*sizeof(R_t), 
            config.cuda_aware ? hipMemcpyDeviceToDevice:hipMemcpyDeviceToHost));
    
        CUFFT_CALL(hipfftCreate(&planR2C));
        CUFFT_CALL(hipfftSetAutoAllocation(planR2C, 0));
        CUFFT_CALL(hipfftMakePlan3d(planR2C, Nx, Ny, Nz, cuFFT<T>::R2Ctype, &ws_r2c));
        CUFFT_CALL(hipfftSetWorkArea(planR2C, in_d));
    
        //Distribute input data
        size_t N1 = Nx/world_size;
        size_t N2 = (Nz/2+1)/world_size;
        size_t send_count = 0;
        size_t recv_count = 0;
        std::vector<size_t> recv_counts;
        std::vector<size_t> ostartz;
        ostartz.push_back(0);
        for (int pidx = 0; pidx < world_size; pidx++){
            size_t Nxpidx = N1 + (pidx<Nx%world_size?1:0);
            size_t Nzpidx = N2 + (pidx<(Nz/2+1)%world_size?1:0);
            recv_req[pidx] = MPI_REQUEST_NULL;
            send_req[pidx] = MPI_REQUEST_NULL;
            ostartz.push_back(ostartz[pidx]+Nzpidx);
    
            //start non-blocking receive for distributed results (asynch to local fft computation)
            MPI_Irecv(&recv_ptr[recv_count], Nx*Ny*Nzpidx*sizeof(C_t), MPI_BYTE, pidx, pidx, MPI_COMM_WORLD, &recv_req[pidx]);
            recv_counts.push_back(recv_count);
            recv_count += Nx*Ny*Nzpidx;
    
            //start non-blocking send for input data
            MPI_Isend(&send_ptr[send_count], Nxpidx*Ny*Nz*sizeof(R_t), MPI_BYTE, pidx, pidx, MPI_COMM_WORLD, &send_req[pidx]);
            send_count += Nxpidx*Ny*Nz;
        }
    
        //wait till all input data has been distributed
        MPI_Waitall(world_size, send_req.data(), MPI_STATUSES_IGNORE);
    
        //compute local fft
        R_t *real    = cuFFT<T>::real(in_d);
        C_t *complex = cuFFT<T>::complex(out_d);
        MPI_Barrier(MPI_COMM_WORLD);
        CUFFT_CALL(cuFFT<T>::execR2C(planR2C, real, complex));
        CUDA_CALL(hipDeviceSynchronize());
    
        CUBLAS_CALL(hipblasCreate(&handle));
    
        int p;
        do {
            // recv_req contains one null handle (i.e. recv_req[pidx_i]) and P1-1 active handles
            // If all active handles are processed, Waitany will return MPI_UNDEFINED
            MPI_Waitany(world_size, recv_req.data(), &p, MPI_STATUSES_IGNORE);
            if (p == MPI_UNDEFINED)
                break;
    
            size_t osizez = N2 + (p<(Nz/2+1)%world_size?1:0);
            
            hipMemcpy3DParms cpy_params = {0};
            cpy_params.srcPos = make_hipPos(0, 0, 0);
            cpy_params.srcPtr = make_hipPitchedPtr(&recv_ptr[recv_counts[p]], osizez*sizeof(C_t), osizez, Ny);
            cpy_params.dstPos = make_hipPos(ostartz[p]*sizeof(C_t), 0, 0);
            cpy_params.dstPtr = make_hipPitchedPtr(res_d, (Nz/2+1)*sizeof(C_t), Nz/2+1, Ny);    
            cpy_params.extent = make_hipExtent(osizez*sizeof(C_t), Ny, Nx);
            cpy_params.kind   = config.cuda_aware ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;   
            
            CUDA_CALL(hipMemcpy3DAsync(&cpy_params));
        } while (p != MPI_UNDEFINED);
        CUDA_CALL(hipDeviceSynchronize());
    
        //compare difference
        Difference_Slab_Z_Then_YX::Difference<T>::difference<<<(Nx*Ny*(Nz/2+1))/1024+1, 1024>>>(complex, res_d, Nx*Ny*(Nz/2+1));
    
        T sum = 0;
        CUBLAS_CALL(Random_Tests<T>::cublasSum(handle, Nx*Ny*(Nz/2+1), complex, 1, &sum));
        CUBLAS_CALL(hipblasDestroy(handle));
    
        std::cout << "Result " << sum << std::endl;
    
        CUFFT_CALL(hipfftDestroy(planR2C));
        MPI_Barrier(MPI_COMM_WORLD);
    }

    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(out_d));
    if (!config.cuda_aware){
        CUDA_CALL(hipFree(res_d));
    } 

    return 0;
}

template <typename T>
int Tests_Slab_Random_Z_Then_YX<T>::compute(const int rank, const int world_size, const int opt, const int runs){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    MPI_Request send_req;
    MPI_Request recv_req;

    size_t N1=Nx/world_size;
    size_t N2=(Nz/2+1)/world_size;
    if (rank < Nx%world_size)
        N1++;
    if (rank < (Nz/2+1)%world_size)
        N2++;

    R_t *in_d, *recv_ptr;
    C_t *out_d, *send_ptr;
    size_t out_size = std::max(N1*Ny*(Nz/2+1), Nx*Ny*N2);

    //allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, N1*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, out_size*sizeof(C_t)));
    
    if (config.cuda_aware){
        recv_ptr = in_d;
        send_ptr = out_d;
    } else {
        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, N1*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*Ny*N2*sizeof(C_t)));
    }

    MPIcuFFT_Slab_Z_Then_YX<T> *mpicuFFT;
    if (opt == 1) 
        mpicuFFT = new MPIcuFFT_Slab_Z_Then_YX_Opt1<T>(config, MPI_COMM_WORLD, world_size);
    else 
        mpicuFFT = new MPIcuFFT_Slab_Z_Then_YX<T>(config, MPI_COMM_WORLD, world_size);
    
    GlobalSize global_size(Nx, Ny, Nz);
    mpicuFFT->initFFT(&global_size, true);

    for (int i = 0; i < runs; i++) {
        //receive input data via MPI
        MPI_Irecv(recv_ptr, N1*Ny*Nz*sizeof(R_t), MPI_BYTE, world_size, rank, MPI_COMM_WORLD, &recv_req);
        MPI_Wait(&recv_req, MPI_STATUSES_IGNORE);
    
        if (!config.cuda_aware){
            CUDA_CALL(hipMemcpyAsync(in_d, recv_ptr, N1*Ny*Nz*sizeof(R_t), hipMemcpyHostToDevice));
            CUDA_CALL(hipDeviceSynchronize());
        }

        MPI_Barrier(MPI_COMM_WORLD);
        //execute
        mpicuFFT->execR2C(out_d, in_d);

        if (!config.cuda_aware){
            CUDA_CALL(hipMemcpyAsync(send_ptr, out_d, Nx*Ny*N2*sizeof(C_t), hipMemcpyDeviceToHost));
            CUDA_CALL(hipDeviceSynchronize());
        }
    
        MPI_Isend(send_ptr, Nx*Ny*N2*sizeof(C_t), MPI_BYTE, world_size, rank, MPI_COMM_WORLD, &send_req);
        MPI_Wait(&send_req, MPI_STATUSES_IGNORE);
        MPI_Barrier(MPI_COMM_WORLD);
    }
    
    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(out_d));

    delete mpicuFFT;
    return 0;
}

template<typename T> 
int Tests_Slab_Random_Z_Then_YX<T>::testcase2(const int opt, const int runs){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;
    int provided;
    hipblasHandle_t handle;
    //initialize MPI
    MPI_Init_thread(NULL, NULL, MPI_THREAD_MULTIPLE, &provided);
    if (provided < MPI_THREAD_MULTIPLE) {
        printf("ERROR: The MPI library does not have full thread support\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    int dev_count;
    CUDA_CALL(hipGetDeviceCount(&dev_count));
    CUDA_CALL(hipSetDevice(rank % dev_count));

    size_t N1=Nx/world_size;
    size_t N2=(Nz/2+1)/world_size;
    if (rank < Nx%world_size)
        N1++;
    if (rank < (Nz/2+1)%world_size)
        N2++;

    R_t *in_d, *inv_d;
    R_t *in_h, *inv_h;
    C_t *out_d;

    size_t out_size = std::max(N1*Ny*(Nz/2+1), Nx*Ny*N2);

    //allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, N1*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&inv_d, N1*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipHostMalloc((void **)&in_h, N1*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipHostMalloc((void **)&inv_h, N1*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, out_size*sizeof(C_t)));
    
    MPIcuFFT_Slab_Z_Then_YX<T> *mpicuFFT;
    if (opt == 1) 
        mpicuFFT = new MPIcuFFT_Slab_Z_Then_YX_Opt1<T>(config, MPI_COMM_WORLD, world_size);
    else 
        mpicuFFT = new MPIcuFFT_Slab_Z_Then_YX<T>(config, MPI_COMM_WORLD, world_size);
        

    GlobalSize global_size(Nx, Ny, Nz);
    mpicuFFT->initFFT(&global_size, true);
    CUBLAS_CALL(hipblasCreate(&handle));
    
    //execute
    for (int i = 0; i < runs; i++){
        this->initializeRandArray(in_d, N1);
        MPI_Barrier(MPI_COMM_WORLD);
        mpicuFFT->execR2C(out_d, in_d);
        MPI_Barrier(MPI_COMM_WORLD);
        mpicuFFT->execC2R(inv_d, out_d);
        MPI_Barrier(MPI_COMM_WORLD);
        //compare difference
        Difference_Slab_Z_Then_YX::Difference<T>::differenceInv<<<(N1*Ny*Nz)/1024+1, 1024>>>(inv_d, in_d, N1*Ny*Nz, Nx*Ny*Nz);
        T sum = 0;
        CUBLAS_CALL(Random_Tests<T>::cublasSumInv(handle, N1*Ny*Nz, inv_d, 1, &sum));
        
        double globalsum = 0;
        double sum_d = static_cast<double>(sum);
        MPI_Allreduce(&sum_d, &globalsum, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
        if (rank == 0)
            std::cout << "Result: " << globalsum << std::endl;
            
    }
    
    CUBLAS_CALL(hipblasDestroy(handle));
    //finalize
    MPI_Finalize();

    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(inv_d));
    CUDA_CALL(hipFree(out_d));

    delete mpicuFFT;
    return 0;
}

template class Tests_Slab_Random_Z_Then_YX<float>;
template class Tests_Slab_Random_Z_Then_YX<double>;


