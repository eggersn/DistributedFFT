#include "hip/hip_runtime.h"
#include "tests_slab_random.hpp"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error at %s:%d\n",__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {             \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)

template<typename T>
int Tests_Slab_Random<T>::initializeRandArray(void* in_d, size_t N){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    hiprandGenerator_t gen;
    R_t *real = cuFFT<T>::real(in_d);

    //create pseudo-random generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    //set seed of generator
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long) clock()));
    CURAND_CALL(Random_Tests<T>::generateUniform(gen, real, N));

    Random_Tests<T>::scaleUniformArray<<<N/1024+1, 1024>>>(real, 255, N);

    CURAND_CALL(hiprandDestroyGenerator(gen));

    return 0;
}

template class Tests_Slab_Random<float>;
template class Tests_Slab_Random<double>;