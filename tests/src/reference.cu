#include "hip/hip_runtime.h"
#include "tests_reference.hpp"
#include "params.hpp"
#include "cufft.hpp"
#include <iostream>
#include <hipfft/hipfft.h>
#include <vector>
#include <thread> 
#include <mutex>
#include <condition_variable>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {             \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)

template<typename T>
int Tests_Reference<T>::initializeRandArray(void* in_d, size_t N1, size_t N2){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    hiprandGenerator_t gen;
    R_t *real = cuFFT<T>::real(in_d);

    //create pseudo-random generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    //set seed of generator
    // CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    //get poisson samples
    CURAND_CALL(Random_Tests<T>::generateUniform(gen, real, N1*N2*Nz));

    Random_Tests<T>::scaleUniformArray<<<(N1*N2*Nz)/1024+1, 1024>>>(real, 255, N1*N2*Nz);

    return 0;
}

template<typename T>
int Tests_Reference<T>::run(const int testcase, const int opt, const int runs) {
    if (Nx == 0)
        throw std::runtime_error("Not initialized");
    if (testcase == 0)
        return this->testcase0(runs);
    else if (testcase == 1)
        return this->testcase1(opt, runs);
    else if (testcase == 2)
        return this->testcase2(opt, runs);
    throw std::runtime_error("Invalid Testcase!");
}

template<typename T>
int Tests_Reference<T>::testcase0(const int runs) {
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    MPI_Init(NULL, NULL);

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    timer = new Timer(MPI_COMM_WORLD, 0, world_size, rank, section_descriptions, "../benchmarks/reference.csv");
    timer->start();

    size_t pidx_i = rank / P2;
    size_t pidx_j = rank % P2;

    R_t *in_d;
    C_t *out_d;

    Partition_Dimensions dim;

    // Determine all Partition_Dimensions
    // dim:
    dim.size_x.resize(P1, Nx/P1);
    for (int i = 0; i < Nx%P1; i++)
        dim.size_x[i]++;
    dim.size_y.resize(P2, Ny/P2);
    for (int j = 0; j < Ny%P2; j++)
        dim.size_y[j]++;
    dim.size_z.resize(1, Nz);
    dim.computeOffsets();
    
    // Generate random input data for each partition
    // Allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, dim.size_x[pidx_i]*dim.size_y[pidx_j]*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, dim.size_x[pidx_i]*dim.size_y[pidx_j]*(Nz/2+1)*sizeof(C_t)));
    
    if (rank == 0) {
        R_t *recv_ptr;
        C_t *send_ptr;

        std::vector<MPI_Request> send_req(world_size, MPI_REQUEST_NULL);
        std::vector<MPI_Request> recv_req(world_size, MPI_REQUEST_NULL);

        R_t *real;
        C_t *complex;
        size_t ws_r2c;
        hipfftHandle planR2C;

        CUDA_CALL(hipMalloc((void **)&real, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipMalloc((void **)&complex, Nx*Ny*(Nz/2+1)*sizeof(C_t)));

        MPI_Barrier(MPI_COMM_WORLD);

        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));

        CUFFT_CALL(hipfftCreate(&planR2C));
        CUFFT_CALL(hipfftMakePlan3d(planR2C, Nx, Ny, Nz, cuFFT<T>::R2Ctype, &ws_r2c));
        CUDA_CALL(hipDeviceSynchronize());
        timer->stop_store("init");
        for (int i = 0; i < runs; i++) {
            this->initializeRandArray(in_d, dim.size_x[pidx_i], dim.size_y[pidx_j]);
            // sync before each run
            MPI_Barrier(MPI_COMM_WORLD);
            timer->start();
            // rank 0 has its input data currently stored at in_d.
            CUDA_CALL(hipMemcpy2DAsync(real, Nz*Ny*sizeof(R_t), in_d, Nz*dim.size_y[0]*sizeof(R_t), Nz*dim.size_y[0]*sizeof(R_t), dim.size_x[0], hipMemcpyDeviceToDevice));
            CUDA_CALL(hipDeviceSynchronize());
    
            // receive the input data from the other ranks
            std::vector<size_t> recv_counts;
            size_t recv_count = 0;
            for (int p_i = 0; p_i < P1; p_i++) {
                for (int p_j = 0; p_j < P2; p_j++) {
                    recv_counts.push_back(recv_count);
                    if (p_i + p_j == 0) {
                        recv_count += dim.size_x[p_i]*dim.size_y[p_j]*Nz;
                    } else {
                        MPI_Irecv(&recv_ptr[recv_count], dim.size_x[p_i]*dim.size_y[p_j]*Nz*sizeof(R_t),
                            MPI_BYTE, p_i*P2+p_j, 0, MPI_COMM_WORLD, &recv_req[p_i*P2+p_j]);
                        recv_count += dim.size_x[p_i]*dim.size_y[p_j]*Nz;
                    }               
    
                }
            }
    
            int p;
            do {
                // recv_req contains one null handle (i.e. recv_req[pidx_i]) and P1-1 active handles
                // If all active handles are processed, Waitany will return MPI_UNDEFINED
                MPI_Waitany(world_size, recv_req.data(), &p, MPI_STATUSES_IGNORE);
    
                if (p == MPI_UNDEFINED)
                    break;
                
                size_t p_i = p / P2;
                size_t p_j = p % P2;
        
                hipMemcpy3DParms cpy_params = {0};
                cpy_params.srcPos = make_hipPos(0, 0, 0);
                cpy_params.srcPtr = make_hipPitchedPtr(&recv_ptr[recv_counts[p]], Nz*sizeof(R_t), Nz, dim.size_y[p_j]);
                cpy_params.dstPos = make_hipPos(0, dim.start_y[p_j], dim.start_x[p_i]);
                cpy_params.dstPtr = make_hipPitchedPtr(real, Nz*sizeof(R_t), Nz, Ny);    
                cpy_params.extent = make_hipExtent(Nz*sizeof(R_t), dim.size_y[p_j], dim.size_x[p_i]);
                cpy_params.kind   = hipMemcpyHostToDevice;   
                
                CUDA_CALL(hipMemcpy3DAsync(&cpy_params));
            } while (p != MPI_UNDEFINED);
            CUDA_CALL(hipDeviceSynchronize());
            timer->stop_store("Finished Receive");
    
            // compute the global 3D FFT
            CUFFT_CALL(cuFFT<T>::execR2C(planR2C, real, complex));
            CUDA_CALL(hipDeviceSynchronize());
            timer->stop_store("3D FFT");

            // redistribute the computed result
            size_t send_count = 0;
            for (int p_i = 0; p_i < P1; p_i++) {
                for (int p_j = 0; p_j < P2; p_j++) {
                    if (p_i + p_j == 0) {
                        // local copy
                        hipMemcpy3DParms cpy_params = {0};
                        cpy_params.srcPos = make_hipPos(0, dim.start_y[p_j], dim.start_x[p_i]);
                        cpy_params.srcPtr = make_hipPitchedPtr(complex, (Nz/2+1)*sizeof(C_t), Nz/2+1, Ny);
                        cpy_params.dstPos = make_hipPos(0, 0, 0);
                        cpy_params.dstPtr = make_hipPitchedPtr(out_d, (Nz/2+1)*sizeof(C_t), Nz/2+1, dim.size_y[p_j]);
                        cpy_params.extent = make_hipExtent((Nz/2+1)*sizeof(C_t), dim.size_y[p_j], dim.size_x[p_i]);
                        cpy_params.kind   = hipMemcpyDeviceToHost;
                        
                        CUDA_CALL(hipMemcpy3DAsync(&cpy_params));

                        send_count += dim.size_x[p_i]*dim.size_y[p_j]*(Nz/2+1);
                    } else {                    
                        hipMemcpy3DParms cpy_params = {0};
                        cpy_params.srcPos = make_hipPos(0, dim.start_y[p_j], dim.start_x[p_i]);
                        cpy_params.srcPtr = make_hipPitchedPtr(complex, (Nz/2+1)*sizeof(C_t), Nz/2+1, Ny);
                        cpy_params.dstPos = make_hipPos(0, 0, 0);
                        cpy_params.dstPtr = make_hipPitchedPtr(&send_ptr[send_count], (Nz/2+1)*sizeof(C_t), Nz/2+1, dim.size_y[p_j]);
                        cpy_params.extent = make_hipExtent((Nz/2+1)*sizeof(C_t), dim.size_y[p_j], dim.size_x[p_i]);
                        cpy_params.kind   = hipMemcpyDeviceToHost;
                        
                        CUDA_CALL(hipMemcpy3DAsync(&cpy_params));
                        CUDA_CALL(hipDeviceSynchronize());
        
                        MPI_Isend(&send_ptr[send_count], dim.size_x[p_i]*dim.size_y[p_j]*(Nz/2+1)*sizeof(C_t), 
                            MPI_BYTE, p_i*P2+p_j, 0, MPI_COMM_WORLD, &send_req[p_i*P2+p_j]);
                        
                        send_count += dim.size_x[p_i]*dim.size_y[p_j]*(Nz/2+1);
                    }
    
                }
            }

            /************************************************************************************
            *
            *      Here, one could continue the previous computation with out_d for rank 0
            *
            *************************************************************************************/            

            MPI_Waitall(world_size, send_req.data(), MPI_STATUSES_IGNORE);
            timer->stop_store("Finished Send");
            timer->stop_store("Run complete");
            timer->gather();

        }
    } else {
        R_t *send_ptr;
        C_t *recv_ptr;

        MPI_Request send_req;
        MPI_Request recv_req;

        if (!cuda_aware) {
            CUDA_CALL(hipHostMalloc((void **)&send_ptr, dim.size_x[pidx_i]*dim.size_y[pidx_j]*Nz*sizeof(R_t)));
            CUDA_CALL(hipHostMalloc((void **)&recv_ptr, dim.size_x[pidx_i]*dim.size_y[pidx_j]*(Nz/2+1)*sizeof(C_t)));
        } else {
            send_ptr = in_d;
            recv_ptr = out_d;
        }
        CUDA_CALL(hipDeviceSynchronize());
        MPI_Barrier(MPI_COMM_WORLD);
        timer->stop_store("init");

        for (int i = 0; i < runs; i++){
            // create new input data for each round (not included in benchmarked time)
            this->initializeRandArray(in_d, dim.size_x[pidx_i], dim.size_y[pidx_j]);
            // sync before each run
            MPI_Barrier(MPI_COMM_WORLD);
            timer->start();
    
            // send input data to rank 0 for global 3D FFT computation
            if (!cuda_aware) 
                CUDA_CALL(hipMemcpy(send_ptr, in_d, dim.size_x[pidx_i]*dim.size_y[pidx_j]*Nz*sizeof(R_t), hipMemcpyDeviceToHost));
    
            MPI_Isend(send_ptr, dim.size_x[pidx_i]*dim.size_y[pidx_j]*Nz*sizeof(R_t),
                MPI_BYTE, 0, 0, MPI_COMM_WORLD, &send_req);
    
            MPI_Wait(&send_req, MPI_STATUS_IGNORE);
            timer->stop_store("Finished Send");
    
            // wait for the computed result to arrive
            MPI_Irecv(recv_ptr, dim.size_x[pidx_i]*dim.size_y[pidx_j]*(Nz/2+1)*sizeof(C_t),
                MPI_BYTE, 0, 0, MPI_COMM_WORLD, &recv_req);
    
            MPI_Wait(&recv_req, MPI_STATUS_IGNORE);
            timer->stop_store("Finished Receive");
    
            if (!cuda_aware)
                CUDA_CALL(hipMemcpy(out_d, recv_ptr, dim.size_x[pidx_i]*dim.size_y[pidx_j]*(Nz/2+1)*sizeof(C_t), hipMemcpyHostToDevice));
            
            /*************************************************************************
            *
            *      Here, one could continue the previous computation with out_d
            *
            **************************************************************************/
            
            timer->stop_store("Run complete");
            timer->gather();
        }
    }
    //finalize
    MPI_Finalize();
    return 0;
}

template<typename T>
int Tests_Reference<T>::testcase1(const int opt, const int runs) {
    using R_t = typename cuFFT<T>::R_t;

    MPI_Init(NULL, NULL);

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    R_t *in_d, *send_ptr, *recv_ptr, *out_d;

    CUDA_CALL(hipMalloc((void **)&in_d, Nx*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, (world_size-1)*Nx*Ny*Nz*sizeof(R_t)));

    if (!cuda_aware) {
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, (world_size-1)*Nx*Ny*Nz*sizeof(R_t)));
    } else {
        send_ptr = in_d;
        recv_ptr = out_d;
    }
    this->initializeRandArray(in_d, Nx, Ny);
    CUDA_CALL(hipDeviceSynchronize());
    MPI_Barrier(MPI_COMM_WORLD);

    std::vector<MPI_Request> send_req(world_size, MPI_REQUEST_NULL);
    std::vector<MPI_Request> recv_req(world_size, MPI_REQUEST_NULL);

    double t1, t2;
    if (opt == 0) {
        for (int i = 0; i < runs+10; i++) {   
            if (i == 10)
                t1 = MPI_Wtime();
            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(send_ptr, in_d, Nx*Ny*Nz*sizeof(R_t), hipMemcpyDeviceToHost));
                CUDA_CALL(hipDeviceSynchronize());
            }
            for (int p = 1; p < world_size; p++) {
                MPI_Isend(send_ptr, Nx*Ny*Nz*sizeof(R_t), MPI_BYTE, (rank+p)%world_size, (rank+p)%world_size, MPI_COMM_WORLD, &send_req[p]);
                MPI_Irecv(&recv_ptr[(p-1)*Nx*Ny*Nz], Nx*Ny*Nz*sizeof(R_t), MPI_BYTE, (rank+p)%world_size, rank, MPI_COMM_WORLD, &recv_req[p]);
            }
            MPI_Waitall(world_size, send_req.data(), MPI_STATUS_IGNORE);
            MPI_Waitall(world_size, recv_req.data(), MPI_STATUS_IGNORE);
            if (!cuda_aware){
                CUDA_CALL(hipMemcpyAsync(out_d, recv_ptr, (world_size-1)*Nx*Ny*Nz*sizeof(R_t), hipMemcpyHostToDevice));
                CUDA_CALL(hipDeviceSynchronize());
            }
        }
        t2 = MPI_Wtime();
    } else if (opt == 1) {
        std::vector<int> sendcounts(world_size, Nx*Ny*Nz*sizeof(R_t));
        sendcounts[rank] = 0;
        std::vector<int> sdispls(world_size, 0);
        std::vector<int> recvcounts(world_size, Nx*Ny*Nz*sizeof(R_t));
        recvcounts[rank] = 0;
        std::vector<int> rdispls(world_size, 0);
        for (int i = 1; i < world_size; i++) {
            rdispls[(rank + i) % world_size] = (i-1)*Nx*Ny*Nz*sizeof(R_t);
        }

        for (int i = 0; i < runs+10; i++) {   
            if (i == 10)
                t1 = MPI_Wtime();
            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(send_ptr, in_d, Nx*Ny*Nz*sizeof(R_t), hipMemcpyDeviceToHost));
                CUDA_CALL(hipDeviceSynchronize());
            }

            MPI_Alltoallv(send_ptr, sendcounts.data(), sdispls.data(), MPI_BYTE, 
                recv_ptr, recvcounts.data(), rdispls.data(), MPI_BYTE, MPI_COMM_WORLD);

            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(out_d, recv_ptr, (world_size-1)*Nx*Ny*Nz*sizeof(R_t), hipMemcpyHostToDevice));
                CUDA_CALL(hipDeviceSynchronize());
            }
        }
        t2 = MPI_Wtime();
    }
    double size = (world_size-1)*Nx*Ny*Nz*sizeof(R_t)*1.0e-6;
    // bandwidth in MB/s
    double bandwidth = size*runs/(t2-t1);
    printf("Bandwidth for rank=%d, runs=%d, size=%f MB in MB/s: %f\n", rank, runs, size, bandwidth);

    MPI_Finalize();
    return 0;
}

struct Callback_Params_Base {
    std::mutex mutex;
    std::condition_variable cv;
    std::vector<int> comm_ready;
};

struct Callback_Params {
    Callback_Params_Base *base_params;

    int p;
};

struct Thread_Params {
    Callback_Params_Base *base_params;

    void* send_ptr;
    int world_size;
    int rank;
    size_t Nx, Ny, Nz;
    std::vector<int> &sizes_x;
    std::vector<int> &sizes_y;
    std::vector<int> &start_y;
};

static void MPIsend_Callback(void *data) {
  struct Callback_Params *params = (Callback_Params *)data;
  struct Callback_Params_Base *base_params = params->base_params;
  {
    std::lock_guard<std::mutex> lk(base_params->mutex);
    base_params->comm_ready.push_back(params->p);
  }
  base_params->cv.notify_one();
}

template <typename T>
static void MPIsend_Thread(Thread_Params &params, std::vector<MPI_Request> &send_req) {
  using R_t = typename cuFFT<T>::R_t;
  struct Callback_Params_Base *base_params = params.base_params;

  R_t *send_ptr = (R_t *) params.send_ptr;

  for (int i = 0; i <params.world_size-1; i++){
    std::unique_lock<std::mutex> lk(base_params->mutex);
    base_params->cv.wait(lk, [base_params]{return !base_params->comm_ready.empty();});

    int p = base_params->comm_ready.back();
    base_params->comm_ready.pop_back();
    size_t oslice = params.Nz*params.start_y[p]*params.sizes_x[params.rank];

    MPI_Isend(&send_ptr[params.Nz*params.start_y[p]*params.sizes_x[params.rank]], 
        params.Nz*params.sizes_y[p]*params.sizes_x[params.rank]*sizeof(R_t), 
        MPI_BYTE, p, p, MPI_COMM_WORLD, &send_req[p]);

    lk.unlock();
  }
}

template<typename T>
int Tests_Reference<T>::testcase2(const int opt, const int runs) {
    using R_t = typename cuFFT<T>::R_t;

    MPI_Init(NULL, NULL);

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    R_t *in_d, *send_ptr, *recv_ptr, *out_d;

    std::vector<int> sizes_x(world_size, Nx/world_size);
    std::vector<int> start_x(world_size, 0);
    for (int i = 0; i < world_size; i++) {
        if (i < Nx % world_size)
            sizes_x[i]++;
        if (i > 0)
            start_x[i] = start_x[i-1] + sizes_x[i-1];
    }

    std::vector<int> sizes_y(world_size, Ny/world_size);
    std::vector<int> start_y(world_size, 0);
    for (int i = 0; i < world_size; i++) {
        if (i < Ny % world_size)
            sizes_y[i]++;
        if (i > 0)
            start_y[i] = start_y[i-1] + sizes_y[i-1];
    }

    CUDA_CALL(hipMalloc((void **)&in_d, sizes_x[rank]*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, Nz*sizes_y[rank]*Nx*sizeof(R_t)));

    if (!cuda_aware) {
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, sizes_x[rank]*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, Nz*sizes_y[rank]*Nx*sizeof(R_t)));
    } else {
        CUDA_CALL(hipMalloc((void **)&send_ptr, sizes_x[rank]*Ny*Nz*sizeof(R_t)));
        recv_ptr = out_d;
    }
    this->initializeRandArray(in_d, sizes_x[rank], Ny);
    CUDA_CALL(hipDeviceSynchronize());
    MPI_Barrier(MPI_COMM_WORLD);

    std::vector<MPI_Request> send_req(world_size, MPI_REQUEST_NULL);
    std::vector<MPI_Request> recv_req(world_size, MPI_REQUEST_NULL);

    double t1, t2;
    if (opt == 0) {
        for (int i = 0; i < runs+10; i++) {   
            if (i == 10)
                t1 = MPI_Wtime();
    
            for (int j = 1; j < world_size; j++) {
                int p = (rank+j)%world_size;
                
                MPI_Irecv(&recv_ptr[Nz*sizes_y[rank]*start_x[p]], Nz*sizes_y[rank]*sizes_x[p]*sizeof(R_t), MPI_BYTE, p, rank, MPI_COMM_WORLD, &recv_req[p]);
    
                CUDA_CALL(hipMemcpy2DAsync(&send_ptr[Nz*start_y[p]*sizes_x[rank]], Nz*sizes_y[p]*sizeof(R_t), 
                    &in_d[Nz*start_y[p]], Nz*Ny*sizeof(R_t), Nz*sizes_y[p]*sizeof(R_t), sizes_x[rank], cuda_aware?hipMemcpyDeviceToDevice:hipMemcpyDeviceToHost));
                CUDA_CALL(hipDeviceSynchronize());
    
                MPI_Isend(&send_ptr[Nz*start_y[p]*sizes_x[rank]], Nz*sizes_y[p]*sizes_x[rank]*sizeof(R_t), MPI_BYTE, p, p, MPI_COMM_WORLD, &send_req[p]);
            }
    
            MPI_Waitall(world_size, send_req.data(), MPI_STATUS_IGNORE);
            MPI_Waitall(world_size, recv_req.data(), MPI_STATUS_IGNORE);
            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(out_d, recv_ptr, Nz*sizes_y[rank]*Nx*sizeof(R_t), hipMemcpyHostToDevice));
                CUDA_CALL(hipDeviceSynchronize());
            }
    
        }
        t2 = MPI_Wtime();
    } else if (opt == 1) {

        std::vector<hipStream_t> streams(world_size);
        CUDA_CALL(hipStreamCreate(&streams[0]));

        Callback_Params_Base base_params;
        std::vector<Callback_Params> params_array;

        for (int i = 1; i < world_size; i++){
            CUDA_CALL(hipStreamCreate(&streams[1]));
            int p = (rank+i)%world_size;
            Callback_Params params = {&base_params, p};
            params_array.push_back(params);
        }

        Thread_Params thread_params = {&base_params, send_ptr, world_size, rank, Nx, Ny, Nz, sizes_x, sizes_y, start_y};

        for (int i = 0; i < runs+10; i++) {   
            if (i == 10)
                t1 = MPI_Wtime();
    
            for (int j = 1; j < world_size; j++) {
                int p = (rank+j)%world_size;
                
                MPI_Irecv(&recv_ptr[Nz*sizes_y[rank]*start_x[p]], Nz*sizes_y[rank]*sizes_x[p]*sizeof(R_t), MPI_BYTE, p, rank, MPI_COMM_WORLD, &recv_req[p]);
    
                CUDA_CALL(hipMemcpy2DAsync(&send_ptr[Nz*start_y[p]*sizes_x[rank]], Nz*sizes_y[p]*sizeof(R_t), 
                    &in_d[Nz*start_y[p]], Nz*Ny*sizeof(R_t), Nz*sizes_y[p]*sizeof(R_t), sizes_x[rank], cuda_aware?hipMemcpyDeviceToDevice:hipMemcpyDeviceToHost, streams[p]));
                CUDA_CALL(hipDeviceSynchronize());
    
                CUDA_CALL(hipLaunchHostFunc(streams[p], MPIsend_Callback, (void *)&params_array[j-1]));
            }
            std::thread mpisend_thread(&MPIsend_Thread<T>, std::ref(thread_params), std::ref(send_req));
            MPI_Waitall(world_size, recv_req.data(), MPI_STATUS_IGNORE);
    
            mpisend_thread.join();
            MPI_Waitall(world_size, send_req.data(), MPI_STATUS_IGNORE);
            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(out_d, recv_ptr, Nz*sizes_y[rank]*Nx*sizeof(R_t), hipMemcpyHostToDevice));
                CUDA_CALL(hipDeviceSynchronize());
            }
    
        }
        t2 = MPI_Wtime();
    } else if (opt == 2) {
        std::vector<MPI_Datatype> MPI_PENCILS(world_size);
        for (int i = 0; i < world_size; i++) {
            MPI_Type_vector(sizes_x[rank], Nz*sizes_y[i]*sizeof(R_t), Nz*Ny*sizeof(R_t), MPI_BYTE, &MPI_PENCILS[i]);
            MPI_Type_commit(&MPI_PENCILS[i]);
        }

        for (int i = 0; i < runs+10; i++) {   
            if (i == 10)
                t1 = MPI_Wtime();
    
            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(send_ptr, in_d, Nz*Ny*sizes_x[rank]*sizeof(R_t), hipMemcpyDeviceToHost));
                CUDA_CALL(hipDeviceSynchronize());
            }

            for (int j = 1; j < world_size; j++) {
                int p = (rank+j)%world_size;
                MPI_Irecv(&recv_ptr[Nz*sizes_y[rank]*start_x[p]], Nz*sizes_y[rank]*sizes_x[p]*sizeof(R_t), MPI_BYTE, p, rank, MPI_COMM_WORLD, &recv_req[p]);
                MPI_Isend(&send_ptr[Nz*start_y[p]], 1, MPI_PENCILS[p], p, p, MPI_COMM_WORLD, &send_req[p]);
            }
            MPI_Waitall(world_size, send_req.data(), MPI_STATUS_IGNORE);
            MPI_Waitall(world_size, recv_req.data(), MPI_STATUS_IGNORE);

            if (!cuda_aware) {
                CUDA_CALL(hipMemcpyAsync(out_d, recv_ptr, Nz*sizes_y[rank]*Nx*sizeof(R_t), hipMemcpyHostToDevice));
                CUDA_CALL(hipDeviceSynchronize());
            }
    
        }
        t2 = MPI_Wtime();
    }

    double size_in = Nz*sizes_y[rank]*(Nx-sizes_x[rank])*sizeof(R_t)*1.0e-6;
    double size_out = Nz*(Ny-sizes_y[rank])*sizes_x[rank]*sizeof(R_t)*1.0e-6;
    // bandwidth in MB/s
    double bandwidth_in = size_in*runs/(t2-t1);
    double bandwidth_out = size_in*runs/(t2-t1);
    printf("Incoming Bandwidth for rank=%d, runs=%d, size=%f MB in MB/s: %f\n", rank, runs, size_in, bandwidth_in);
    printf("Outgoing Bandwidth for rank=%d, runs=%d, size=%f MB in MB/s: %f\n", rank, runs, size_out, bandwidth_out);

    MPI_Finalize();
    return 0;
}

template class Tests_Reference<float>;
template class Tests_Reference<double>;