#include "hip/hip_runtime.h"
/* 
* Copyright (C) 2021 Simon Egger
* 
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
* 
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
* 
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "tests_base.hpp"

__global__ void scaleUniformArrayFloat(cuFFT<float>::R_t* data_d, cuFFT<float>::R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

__global__ void scaleUniformArrayDouble(cuFFT<double>::R_t* data_d, cuFFT<double>::R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

decltype(hiprandGenerateUniformDouble)* Random_Tests<double>::generateUniform = hiprandGenerateUniformDouble;
decltype(hipblasDzasum)* Random_Tests<double>::cublasSum = hipblasDzasum;
decltype(hipblasDasum)* Random_Tests<double>::cublasSumInv = hipblasDasum;
decltype(hipblasIdamax)* Random_Tests<double>::cublasMaxIndex = hipblasIdamax;
decltype(scaleUniformArrayDouble)* Random_Tests<double>::scaleUniformArray = scaleUniformArrayDouble;