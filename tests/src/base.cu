#include "hip/hip_runtime.h"
#include "tests_base.hpp"

__global__ void scaleUniformArrayFloat(cuFFT<float>::R_t* data_d, cuFFT<float>::R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

__global__ void scaleUniformArrayDouble(cuFFT<double>::R_t* data_d, cuFFT<double>::R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

decltype(hiprandGenerateUniformDouble)* Random_Tests<double>::generateUniform = hiprandGenerateUniformDouble;
decltype(hipblasDzasum)* Random_Tests<double>::cublasSum = hipblasDzasum;
decltype(hipblasDasum)* Random_Tests<double>::cublasSumInv = hipblasDasum;
decltype(hipblasIdamax)* Random_Tests<double>::cublasMaxIndex = hipblasIdamax;
decltype(scaleUniformArrayDouble)* Random_Tests<double>::scaleUniformArray = scaleUniformArrayDouble;