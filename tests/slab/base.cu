#include "hip/hip_runtime.h"
#include "tests_slab_random.hpp"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error at %s:%d\n",__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {             \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)

__global__ void scaleUniformArrayFloat(cuFFT<float>::R_t* data_d, cuFFT<float>::R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

__global__ void scaleUniformArrayDouble(cuFFT<double>::R_t* data_d, cuFFT<double>::R_t factor, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        data_d[i] *= factor;
    }
}

decltype(hiprandGenerateUniformDouble)* Random_Tests<double>::generateUniform = hiprandGenerateUniformDouble;
decltype(hipblasDzasum)* Random_Tests<double>::cublasSum = hipblasDzasum;
decltype(scaleUniformArrayDouble)* Random_Tests<double>::scaleUniformArray = scaleUniformArrayDouble;

template<typename T>
int Tests_Slab_Random<T>::initializeRandArray(void* in_d, size_t N1){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    hiprandGenerator_t gen;
    R_t *real = cuFFT<T>::real(in_d);

    //create pseudo-random generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    //set seed of generator
    // CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    //get poisson samples
    CURAND_CALL(Random_Tests<T>::generateUniform(gen, real, N1*Ny*Nz));

    Random_Tests<T>::scaleUniformArray<<<(N1*Ny*Nz)/1024+1, 1024>>>(real, 255, N1*Ny*Nz);

    return 0;
}

template class Tests_Slab_Random<float>;
template class Tests_Slab_Random<double>;