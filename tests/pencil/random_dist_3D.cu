#include "hip/hip_runtime.h"
#include "mpicufft_pencil.hpp"
#include "tests_pencil_random_3d.hpp"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0) 
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error at %s:%d\n",__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {             \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)

namespace Difference_Pencil_3D {
    // Difference
    // Definition in tests/pencil/base.cu
    __global__ void differenceFloat(cuFFT<float>::C_t* array1, cuFFT<float>::C_t* array2, int n){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) {
            array1[i].x -= array2[i].x;
            array1[i].y -= array2[i].y;
        }
    }
    
    __global__ void differenceDouble(cuFFT<double>::C_t* array1, cuFFT<double>::C_t* array2, int n){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) {
            array1[i].x -= array2[i].x;
            array1[i].y -= array2[i].y;
        }
    }
    
    template<typename T> 
    struct Difference { 
        static decltype(differenceFloat)* difference;
    };
    template<typename T> decltype(differenceFloat)* Difference<T>::difference = differenceFloat;
    
    template<> struct Difference<double> { 
        static decltype(differenceDouble)* difference;
    };
    decltype(differenceDouble)* Difference<double>::difference = differenceDouble;
}

template<typename T> 
int Tests_Pencil_Random_3D<T>::run(const int testcase, const int opt, const int runs){
    if (testcase == 0)
        return this->testcase0(opt, runs);
    else if (testcase == 1)
        return this->testcase1(opt, runs);
    return -1;
}

template<typename T> 
int Tests_Pencil_Random_3D<T>::testcase0(const int opt, const int runs){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    //initialize MPI
    MPI_Init(NULL, NULL);

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    size_t pidx_i = rank / P2;
    size_t pidx_j = rank % P2;
        
    //initialize MPIcuFFT
    MPIcuFFT_Pencil<T> mpicuFFT(MPI_COMM_WORLD, true);

    Pencil_Partition partition(P1, P2);
    GlobalSize global_size(Nx, Ny, Nz);
    mpicuFFT.initFFT(&global_size, &partition, true);

    // Allocate Memory
    Partition_Dimensions input_dim, transposed_dim, output_dim;
    mpicuFFT.getPartitionDimensions(input_dim, transposed_dim, output_dim);

    size_t out_size = std::max(input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*(Nz/2+1), transposed_dim.size_x[pidx_i]*transposed_dim.size_y[0]*transposed_dim.size_z[pidx_j]);
    out_size = std::max(out_size, output_dim.size_x[0]*output_dim.size_y[pidx_i]*output_dim.size_z[pidx_j]);

    R_t *in_d;
    C_t *out_d;
    // C_t *out_h;

    CUDA_CALL(hipMalloc((void **)&in_d, input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, out_size*sizeof(C_t)));
    //allocate memory (host)
    // out_h = (T *)calloc(out_size, sizeof(C_t));

    for (int i = 0; i < runs; i++) {
        this->initializeRandArray(in_d, input_dim.size_x[pidx_i], input_dim.size_y[pidx_j]);
        MPI_Barrier(MPI_COMM_WORLD);
        mpicuFFT.execR2C(out_d, in_d);
        MPI_Barrier(MPI_COMM_WORLD);
    }

    // CUDA_CALL(hipMemcpy(out_h, out_d, out_size*sizeof(C_t), hipMemcpyDeviceToHost));

    //do stuff with out_h / out_d

    //finalize
    MPI_Finalize();

    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(out_d));
    // free(out_h);

    return 0;
}

template<typename T>
int Tests_Pencil_Random_3D<T>::testcase1(const int opt, const int runs) {      
    //initialize MPI
    MPI_Init(NULL, NULL);

    //number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    //get global rank
    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == world_size-1){
        this->coordinate(world_size, runs);
    } else{
        this->compute(rank, world_size-1, opt, runs);
    }
    
    //finalize
    MPI_Finalize();

    return 0;
}

template<typename T>
int Tests_Pencil_Random_3D<T>::coordinate(const int world_size, const int runs) {
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    std::vector<MPI_Request> send_req;
    std::vector<MPI_Request> recv_req;
    
    R_t *in_d, *send_ptr;
    C_t *out_d, *recv_ptr, *res_d;
    
    Partition_Dimensions input_dim;
    Partition_Dimensions transposed_dim;
    Partition_Dimensions output_dim;

    send_req.resize(world_size, MPI_REQUEST_NULL);
    recv_req.resize(world_size, MPI_REQUEST_NULL);

    // Determine all Partition_Dimensions
    // input_dim:
    input_dim.size_x.resize(P1, Nx/P1);
    for (int i = 0; i < Nx%P1; i++)
        input_dim.size_x[i]++;
    input_dim.size_y.resize(P2, Ny/P2);
    for (int j = 0; j < Ny%P2; j++)
        input_dim.size_y[j]++;
    input_dim.size_z.resize(1, Nz);
    input_dim.computeOffsets();
    // transposed_dim:
    transposed_dim.size_x = input_dim.size_x;
    transposed_dim.size_y.resize(1, Ny);
    transposed_dim.size_z.resize(P2, (Nz/2+1)/P2);
    for (int k = 0; k < (Nz/2+1)%P2; k++)
        transposed_dim.size_z[k]++;
    transposed_dim.computeOffsets();
    // output_dim:
    output_dim.size_x.resize(1, Nx);
    output_dim.size_y.resize(P1, Ny/P1);
    for (int j = 0; j < Ny%P1; j++)
        output_dim.size_y[j]++;
    output_dim.size_z = transposed_dim.size_z;
    output_dim.computeOffsets();

    // Generate random input data for each partition
    // Allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, Nx*Ny*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    CUDA_CALL(hipMalloc((void **)&res_d, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    
    if (cuda_aware == 1){
        CUDA_CALL(hipMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipMalloc((void **)&recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    } else {
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, Nx*Ny*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, Nx*Ny*(Nz/2+1)*sizeof(C_t)));
    }

    for (int i = 0; i < runs; i++) {
        //random initialization of full Nx*Ny*Nz array
        this->initializeRandArray(in_d, Nx, Ny);
    
        std::vector<size_t> recv_counts;
        size_t recv_count = 0;
        size_t send_count = 0;
        for (size_t p_i = 0; p_i < P1; p_i++){
            for (size_t p_j = 0; p_j < P2; p_j++){
                hipMemcpy3DParms cpy_params = {0};
                cpy_params.srcPos = make_hipPos(0, input_dim.start_y[p_j], input_dim.start_x[p_i]);
                cpy_params.srcPtr = make_hipPitchedPtr(in_d, Nz*sizeof(R_t), Nz, Ny);
                cpy_params.dstPos = make_hipPos(0, 0, 0);
                cpy_params.dstPtr = make_hipPitchedPtr(&send_ptr[send_count], Nz*sizeof(R_t), Nz, input_dim.size_y[p_j]);
                cpy_params.extent = make_hipExtent(Nz*sizeof(R_t), input_dim.size_y[p_j], input_dim.size_x[p_i]);
                cpy_params.kind   = cuda_aware==1 ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost;
                
                CUDA_CALL(hipMemcpy3DAsync(&cpy_params));
                CUDA_CALL(hipDeviceSynchronize());
                
                recv_counts.push_back(recv_count);
    
                //start non-blocking receive for distributed results (asynch to local fft computation)
                MPI_Irecv(&recv_ptr[recv_count], output_dim.size_x[0]*output_dim.size_y[p_i]*output_dim.size_z[p_j]*sizeof(C_t), 
                MPI_BYTE, p_i*P2+p_j, world_size, MPI_COMM_WORLD, &recv_req[p_i*P2+p_j]);
                recv_count += output_dim.size_x[0] * output_dim.size_y[p_i] * output_dim.size_z[p_j];
                
                //start non-blocking send for input data
                MPI_Isend(&send_ptr[send_count], input_dim.size_x[p_i]*input_dim.size_y[p_j]*Nz*sizeof(R_t), 
                MPI_BYTE, p_i*P2+p_j, world_size, MPI_COMM_WORLD, &send_req[p_i*P2+p_j]);
                
                send_count += input_dim.size_x[p_i] * input_dim.size_y[p_j] * Nz;
            }
        }
    
        MPI_Waitall(world_size-1, send_req.data(), MPI_STATUSES_IGNORE);
        
        // compute full fft locally
        size_t ws_r2c;
        
        hipfftHandle planR2C;
        hipblasHandle_t handle;
        
        R_t *real    = cuFFT<T>::real(in_d);
        C_t *complex = cuFFT<T>::complex(out_d);
        
        CUFFT_CALL(hipfftCreate(&planR2C));
        CUFFT_CALL(hipfftMakePlan3d(planR2C, Nx, Ny, Nz, cuFFT<T>::R2Ctype, &ws_r2c));
        
        MPI_Barrier(MPI_COMM_WORLD);
        CUFFT_CALL(cuFFT<T>::execR2C(planR2C, real, complex));
        CUDA_CALL(hipDeviceSynchronize());
    
        CUBLAS_CALL(hipblasCreate(&handle));
        
    
        int p;
        do {
            // recv_req contains one null handle (i.e. recv_req[pidx_i]) and P1-1 active handles
            // If all active handles are processed, Waitany will return MPI_UNDEFINED
            MPI_Waitany(world_size-1, recv_req.data(), &p, MPI_STATUSES_IGNORE);
            if (p == MPI_UNDEFINED)
                break;
            
            size_t p_i = p / P2;
            size_t p_j = p % P2;
    
            hipMemcpy3DParms cpy_params = {0};
            cpy_params.srcPos = make_hipPos(0, 0, 0);
            cpy_params.srcPtr = make_hipPitchedPtr(&recv_ptr[recv_counts[p]], output_dim.size_z[p_j]*sizeof(C_t), output_dim.size_z[p_j], output_dim.size_y[p_i]);
            cpy_params.dstPos = make_hipPos(output_dim.start_z[p_j]*sizeof(C_t), output_dim.start_y[p_i], 0);
            cpy_params.dstPtr = make_hipPitchedPtr(res_d, (Nz/2+1)*sizeof(C_t), Nz/2+1, Ny);    
            cpy_params.extent = make_hipExtent(output_dim.size_z[p_j]*sizeof(C_t), output_dim.size_y[p_i], output_dim.size_x[0]);
            cpy_params.kind   = cuda_aware==1 ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;   
            
            CUDA_CALL(hipMemcpy3DAsync(&cpy_params));
        } while (p != MPI_UNDEFINED);
        CUDA_CALL(hipDeviceSynchronize());
        
        Difference_Pencil_3D::Difference<T>::difference<<<(Nx*Ny*(Nz/2+1))/1024+1, 1024>>>(complex, res_d, Nx*Ny*(Nz/2+1));
    
        T sum = 0;
        CUBLAS_CALL(Random_Tests<T>::cublasSum(handle, Nx*Ny*(Nz/2+1), complex, 1, &sum));

        printf("\nResults: %f\n", sum);
    
        CUBLAS_CALL(hipblasDestroy(handle));
        
        CUFFT_CALL(hipfftDestroy(planR2C));
        MPI_Barrier(MPI_COMM_WORLD);
    }

    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(out_d));
    CUDA_CALL(hipFree(res_d));
    if (cuda_aware == 1){
        CUDA_CALL(hipFree(send_ptr));
        CUDA_CALL(hipFree(recv_ptr));
    } else {
        CUDA_CALL(hipHostFree(send_ptr));
        CUDA_CALL(hipHostFree(recv_ptr));
    }

    return 0;    
}

template<typename T>
int Tests_Pencil_Random_3D<T>::compute(const int rank, const int world_size, const int opt, const int runs){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    MPI_Request send_req;
    MPI_Request recv_req;

    R_t *in_d, *recv_ptr;
    C_t *out_d, *send_ptr;

    size_t pidx_i = rank / P2;
    size_t pidx_j = rank % P2;

    //initialize MPIcuFFT
    MPIcuFFT_Pencil<T> mpicuFFT(MPI_COMM_WORLD, cuda_aware==1, world_size);

    Pencil_Partition partition(P1, P2);
    GlobalSize global_size(Nx, Ny, Nz);
    mpicuFFT.initFFT(&global_size, &partition, true);

    // Allocate Memory
    Partition_Dimensions input_dim, transposed_dim, output_dim;
    mpicuFFT.getPartitionDimensions(input_dim, transposed_dim, output_dim);

    size_t out_size = std::max(input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*(Nz/2+1), transposed_dim.size_x[pidx_i]*transposed_dim.size_y[0]*transposed_dim.size_z[pidx_j]);
    out_size = std::max(out_size, output_dim.size_x[0]*output_dim.size_y[pidx_i]*output_dim.size_z[pidx_j]);

    //allocate memory (device)
    CUDA_CALL(hipMalloc((void **)&in_d, input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*Nz*sizeof(R_t)));
    CUDA_CALL(hipMalloc((void **)&out_d, out_size*sizeof(C_t)));
    
    if (cuda_aware == 1){
        recv_ptr = in_d;
        send_ptr = out_d;
    } else {
        CUDA_CALL(hipHostMalloc((void **)&recv_ptr, input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*Nz*sizeof(R_t)));
        CUDA_CALL(hipHostMalloc((void **)&send_ptr, output_dim.size_x[0]*output_dim.size_y[pidx_i]*output_dim.size_z[pidx_j]*sizeof(C_t)));
    }

    for (int i = 0; i < runs; i++) {
        //receive input data via MPI
        MPI_Irecv(recv_ptr, input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*Nz*sizeof(R_t), MPI_BYTE, world_size, world_size+1, MPI_COMM_WORLD, &recv_req);
        MPI_Wait(&recv_req, MPI_STATUSES_IGNORE);
    
        if (cuda_aware == 0){
            CUDA_CALL(hipMemcpyAsync(in_d, recv_ptr, input_dim.size_x[pidx_i]*input_dim.size_y[pidx_j]*Nz*sizeof(R_t), hipMemcpyHostToDevice));
        }
    
        MPI_Barrier(MPI_COMM_WORLD);
        CUDA_CALL(hipDeviceSynchronize());
    
        //execute
        
        mpicuFFT.execR2C(out_d, in_d);
    
        if (cuda_aware == 0){
            CUDA_CALL(hipMemcpy(send_ptr, out_d, output_dim.size_x[0]*output_dim.size_y[pidx_i]*output_dim.size_z[pidx_j]*sizeof(C_t), hipMemcpyDeviceToHost));
        }
    
        MPI_Isend(send_ptr, output_dim.size_x[0]*output_dim.size_y[pidx_i]*output_dim.size_z[pidx_j]*sizeof(C_t), MPI_BYTE, world_size, world_size+1, MPI_COMM_WORLD, &send_req);
        MPI_Wait(&send_req, MPI_STATUSES_IGNORE);
        MPI_Barrier(MPI_COMM_WORLD);
        
    }
    
    CUDA_CALL(hipFree(in_d));
    CUDA_CALL(hipFree(out_d));

    if (cuda_aware == 0) {
        CUDA_CALL(hipHostFree(recv_ptr));
        CUDA_CALL(hipHostFree(send_ptr));
    }

    return 0;
}

template class Tests_Pencil_Random_3D<float>;
template class Tests_Pencil_Random_3D<double>;

