#include "tests_pencil_random.hpp"
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {    \
    printf("Error at %s:%d\n",__FILE__,__LINE__);               \
    return EXIT_FAILURE;}} while(0)
#define CUBLAS_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) {    \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)
#define CUFFT_CALL(x) do { if((x)!=HIPFFT_SUCCESS) {             \
    printf("Error %d at %s:%d\n",x,__FILE__,__LINE__);          \
    return EXIT_FAILURE;}} while(0)

template<typename T>
int Tests_Pencil_Random<T>::initializeRandArray(void* in_d, size_t N1, size_t N2){
    using R_t = typename cuFFT<T>::R_t;
    using C_t = typename cuFFT<T>::C_t;

    hiprandGenerator_t gen;
    R_t *real = cuFFT<T>::real(in_d);

    //create pseudo-random generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    //set seed of generator
    // CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    //get poisson samples
    CURAND_CALL(Random_Tests<T>::generateUniform(gen, real, N1*N2*Nz));

    Random_Tests<T>::scaleUniformArray<<<(N1*N2*Nz)/1024+1, 1024>>>(real, 255, N1*N2*Nz);

    return 0;
}

template class Tests_Pencil_Random<float>;
template class Tests_Pencil_Random<double>;